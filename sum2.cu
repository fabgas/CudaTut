
#include <hip/hip_runtime.h>
/*
 * JCuda - Java bindings for NVIDIA CUDA driver and runtime API
 * http://www.jcuda.org
 *
 *
 * This code is based on the NVIDIA 'reduction' CUDA sample,
 * Copyright 1993-2010 NVIDIA Corporation.
 */
extern "C"
__global__ void sum(float *g_idata,float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[]; 
    unsigned int tid = threadIdx.x; // thread courant dans le block
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; // index g�n�ral
	sdata[tid] = g_idata[i]; // copy vers la shared memory du block
	__syncthreads(); // on attends tous les blocks
	
	if (i >= n) return; // on coupe au dela du cutoff
	// do reduction in shared mem for one block 
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;
		if (index<blockDim.x) { // si correspond � un multiple de la dimension
		sdata[index] += sdata[index + s];
		}
		__syncthreads(); // on attends
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0]; 
	}