
#include <hip/hip_runtime.h>
/*
 * JCuda - Java bindings for NVIDIA CUDA driver and runtime API
 * http://www.jcuda.org
 *
 *
 * This code is based on the NVIDIA 'reduction' CUDA sample,
 * Copyright 1993-2010 NVIDIA Corporation.
 */
extern "C"
__global__ void threads(float *g_idata,unsigned int n)
{
   

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;
	if (i<n) {
		printf("Hello world %d - %d : %f\n",tid,i,g_idata[i]);
	}
}